#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <typeinfo>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include <mpi.h>
#include <nccl.h>

int me, np;
ncclComm_t NCCL_COMM_WORLD;
hipblasHandle_t cublas_handle;

void check(hipError_t rc)
{   
    if (rc!=hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorName(rc) << "=" << hipGetErrorString(rc) << std::endl;
        std::abort();
    }
}

void check(hiprandStatus_t rc)
{   
    if (rc!=HIPRAND_STATUS_SUCCESS) {
        std::cerr << "CURAND error: " << rc << std::endl;
        std::abort();
    }
}
void check(ncclResult_t rc)
{
    if (rc != ncclSuccess) {
        std::cerr << "NCCL error: " << ncclGetErrorString(rc) << std::endl;
        std::abort();
    }
}

void check(hipblasStatus_t rc)
{
    if (rc!=HIPBLAS_STATUS_SUCCESS) {
#if defined(CUBLAS_VERSION) && (CUBLAS_VERSION >= (11*10000+4*100+2))
        std::cerr << "CUBLAS error: " << cublasGetStatusName(rc) << "=" << cublasGetStatusString(rc) << std::endl;
#else
        std::cerr << "CUBLAS error: " << rc << std::endl;
#endif
        std::abort();
    }
}

template <typename T>
ncclDataType_t get_NCCL_Datatype(T t) {
    std::cerr << "NCCL datatype resolution failed for type " << typeid(T).name() << std::endl;
    std::abort();
}

template <>
constexpr ncclDataType_t get_NCCL_Datatype(double d) { return ncclFloat64; }
template <>
constexpr ncclDataType_t get_NCCL_Datatype(float d) { return ncclFloat32; }
template <>
constexpr ncclDataType_t get_NCCL_Datatype(half d) { return ncclFloat16; }
template <>
constexpr ncclDataType_t get_NCCL_Datatype(hip_bfloat16 d) { return ncclBfloat16; }
template <>
constexpr ncclDataType_t get_NCCL_Datatype(int i) { return ncclInt32; }


#ifdef __NVCC__

template <typename T>
__global__
void cast_from_double(T * __restrict__ out, const double * __restrict__ in, unsigned n)
{
    const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;         
    if (i < n) {
        out[i] = (T)in[i];
    }
}

template <typename T>
__global__
void cast_to_double(double * __restrict__ out, const T * __restrict__ in, unsigned n)
{
    const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;         
    if (i < n) {
        out[i] = (double)in[i];
    }
}

template <typename T>
__global__
void scale(T * __restrict__ out, int s, unsigned n)
{
    const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;         
    if (i < n) {
        out[i] *= s;
    }
}

template <typename T>
__global__
void diff(double * __restrict__ out, const T * __restrict__ in, const double * __restrict__ ref, unsigned n)
{
    const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;         
    if (i < n) {
        out[i] = ref[i] - (double)in[i];
    }
}

#endif

template <typename T> 
void print_norm(const T * x, int n, const std::string & name)
{}

template<>
void print_norm(const float * x, int n, const std::string & name)
{
    float result;
    check( hipblasSnrm2(cublas_handle, n, x, 1, &result) );
    std::cout << me << ": " << "the 2-norm of " << name << " is " << result << std::endl;
}

template<>
void print_norm(const double * x, int n, const std::string & name)
{
    double result;
    check( hipblasDnrm2(cublas_handle, n, x, 1, &result) );
    std::cout << me << ": " << "the 2-norm of " << name << " is " << result << std::endl;
}

template <typename T>
void reduce_test(int count)
{
    const size_t bytes = count * sizeof(T);

    const unsigned threads_per_block = 256;
    const unsigned blocks_per_grid = (count + threads_per_block - 1) / threads_per_block;

    hiprandGenerator_t gen;
    check( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
    check( hiprandSetPseudoRandomGeneratorSeed(gen, me * 1234ULL) );
    check( hipDeviceSynchronize() );

    double * ref = nullptr;
    check( hipMalloc((void**)&ref, count * sizeof(double)) );
    check( hiprandGenerateUniformDouble(gen, ref, count) );
    scale<<<blocks_per_grid, threads_per_block>>>(ref, 10, count);
    check( hipDeviceSynchronize() );
    print_norm(ref, count, "ref");

    double * res = nullptr;
    check( hipMalloc((void**)&res, count * sizeof(double)) );
    check( hipMemset((void*)res, 0, count * sizeof(double)) );
    check( hipDeviceSynchronize() );
    //print_norm(res, count, "res");

    {
        T * in  = nullptr;
        check( hipMalloc((void**)&in,  bytes) );
        //check( hipMemset((void*)in, 0xFFFFFFFF, bytes) );
        cast_from_double<<<blocks_per_grid, threads_per_block>>>(in, ref, count);
        check( hipDeviceSynchronize() );
        print_norm(in, count, "in");

        T * out = nullptr;
        check( hipMalloc((void**)&out, bytes) );
        check( hipMemset((void*)out, 0, bytes) );
        check( hipDeviceSynchronize() );

        check( ncclAllReduce(in, out, count, get_NCCL_Datatype(*in), ncclSum, NCCL_COMM_WORLD, 0 /* default stream */) );
        check( hipDeviceSynchronize() );
        if (me == 0) print_norm(out, count, "out");

        check( ncclAllReduce(ref, ref, count, ncclDouble, ncclSum, NCCL_COMM_WORLD, 0 /* default stream */) );
        check( hipDeviceSynchronize() );
        if (me == 0) print_norm(ref, (int)count, "ref (after ncclAllReduce)");

        diff<<<blocks_per_grid, threads_per_block>>>(res, out, ref, count);

        double result;
        check( hipblasDnrm2(cublas_handle, (int)count, res, 1, &result) );
        if (me == 0) {
            std::cout << me << ": difference between " << typeid(T).name() <<" and double is " << result << std::endl;
        }

        check( hipFree((void*)out) );
        check( hipFree((void*)in) );
    }

    check( hipFree((void*)res) );
    check( hipFree((void*)ref) );
    check( hiprandDestroyGenerator(gen) );
}

int main(int argc, char* argv[])
{
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &me);
    MPI_Comm_size(MPI_COMM_WORLD, &np);

    const int count = (argc > 1) ? std::atoi(argv[1]) : 1024*1024;
    if (me == 0) {
        std::cout << "count = " << count << std::endl;
    }

    int num_gpus;
    check( hipGetDeviceCount(&num_gpus) );
    if (np > num_gpus) {
        std::cerr << "run with no more MPI processes than GPUs" << std::endl;
        MPI_Abort(MPI_COMM_WORLD,num_gpus);
    }
    check( hipSetDevice(me % num_gpus) );
    MPI_Barrier(MPI_COMM_WORLD);

    check( hipblasCreate(&cublas_handle) );

    ncclUniqueId uniqueId;
    if (me == 0) {
        check( ncclGetUniqueId(&uniqueId) );
    }
    MPI_Bcast(&uniqueId, sizeof(uniqueId), MPI_BYTE, 0, MPI_COMM_WORLD);

    check( ncclGroupStart() );
    check( ncclCommInitRank(&NCCL_COMM_WORLD, np, uniqueId, me) );
    check( ncclGroupEnd() );
    MPI_Barrier(MPI_COMM_WORLD);

    reduce_test<float>(count);
    MPI_Barrier(MPI_COMM_WORLD);

    reduce_test<half>(count);
    MPI_Barrier(MPI_COMM_WORLD);

    reduce_test<hip_bfloat16>(count);
    MPI_Barrier(MPI_COMM_WORLD);

    check( ncclCommDestroy(NCCL_COMM_WORLD) );

    if (me == 0) std::cout << "FINISHED" << std::endl;

    MPI_Finalize();
    return 0;
}
